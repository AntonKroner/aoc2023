#include <vector>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template<typename T> class Vector {
  private:
    __device__ T* elements;
    __device__ size_t length;
  public:
    Vector(T element) {
      this->length = 1;
      hipMalloc(this.elements, sizeof(T));
      this->elements[0] = element;
    }
    Vector(size_t length, T* elements) {
      this->length = length;
      hipMalloc(this->elements, sizeof(T) * length);
      hipMemcpy(this->elements, elements, sizeof(T) * length, hipMemcpyHostToDevice);
    }
    Vector(std::vector<T> vector) {
      this->length = vector.size();
      hipMalloc(this->elements, sizeof(T) * this->length);
      hipMemcpy(
              this->elements,
              vector.data(),
              sizeof(T) * this->length,
              hipMemcpyHostToDevice
      );
    }
    ~vectorClass() {
      hipFree(this->elements);
    }
    __host__ size_t push(size_t length, T* elements) {
      T* newElements;
      hipMalloc(newElements, sizeof(T) * (this->length + length));
      hipMemcpy(
              newElements,
              this->elements,
              sizeof(T) * this->length,
              hipMemcpyHostToDevice
      );
      hipMemcpy(
              &newElements[this->length],
              elements,
              sizeof(T) * length,
              hipMemcpyHostToDevice
      );
      hipFree(this->elements);
      this->elements = newElements;
      this->length += length;
    }
    template<typename R> __device__ Vector<R> map(
            R (*mapper)(T),
            size_t thread,
            size_t threads,
            R (&shared)[this->length]
    ) {
      R result[this->length];
      for (size_t i = 0; this->length > i; i++) {
        result[i] = mapper(this->elements[i]);
      }
      return Vector<R>(this->length, result);
    }
};
