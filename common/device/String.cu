#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <string>

class String {
  private:
    __device__ char* characters;
    __device__ size_t length;
    String(std::string string) {
      const size_t length = string.size();
      hipMalloc(&characters, sizeof(char) * length);
      hipMemcpy(
              this->characters,
              string.data(),
              sizeof(char) * length,
              hipMemcpyHostToDevice
      );
      hipMemcpy(&(this->length), &length, sizeof(size_t), hipMemcpyHostToDevice);
    }
  public:
    __device__ __inline_hint__ size_t length() {
      return this->length;
    }
    __host__ std::string toHost() {
      size_t length = 0;
      hipMemcpy(&(length), &(this->length), sizeof(size_t), hipMemcpyDeviceToHost);
      char* characters = (char*)malloc(sizeof(char) * length);
      hipMemcpy(
              characters,
              this->characters,
              sizeof(char) * length,
              hipMemcpyDeviceToHost
      );
      std::string result = std::string(characters, length);
      free(characters);
      return result;
    }

    __host__ static String open(std::string string) {
      return String(string);
    }
};
