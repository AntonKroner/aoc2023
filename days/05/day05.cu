#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <string>
#include <stdlib.h>
#include <algorithm>

__host__ __device__ static inline bool isNumber(const char character) {
	return character >= '0' && character <= '9';
}
__device__ static size_t findDestination(
	const size_t length,
	const size_t sources[],
	const size_t destinations[],
	const size_t ranges[],
	const size_t input
) {
	size_t result = 0;
	for (size_t i = 0; length > i; i++) {
		if (input > sources[i] && sources[i] + ranges[i] > input) {
			result = destinations[i] - sources[i] + input;
			break;
		}
	}
	return result ? result : input;
}
__global__ static void findLocations(
	const size_t* seeds,
	const size_t length,
	const size_t* lengths,
	const size_t* sources,
	const size_t* destinations,
	const size_t* ranges,
	size_t* locations
) {
	size_t result = seeds[threadIdx.x];
	for (size_t i = 0; (length - 1) > i; i++) {
		result = findDestination(lengths[i + 1] - lengths[i], &sources[lengths[i]], &destinations[lengths[i]], &ranges[lengths[i]], result);
	}
	locations[threadIdx.x] = result;
}
std::vector<std::string> splitString(const std::string& string, char delimiter) {
	std::vector<std::string> result;
	size_t start = 0;
	size_t end = string.find(delimiter);
	while (end != std::string::npos) {
		std::string token = string.substr(start, end - start);
		if (!token.empty()) {
			result.push_back(token);
		}
		start = end + 1;
		end = string.find(delimiter, start);
	}
	std::string lastToken = string.substr(start);
	if (!lastToken.empty()) {
		result.push_back(lastToken);
	}
	return result;
}
void day05(const size_t part) {
	std::string line;
	std::ifstream input;
	std::vector<std::string> lines;
	input.open("./days/05/input.txt");
	if (input.is_open()) {
		while (std::getline(input, line)) {
			lines.push_back(line);
		}
		input.close();
	}
	else {
		std::cout << "file did not open" << std::endl;
	}
	std::vector<size_t> seeds;
	std::vector<size_t> lengths;
	std::vector<size_t> sources;
	std::vector<size_t> ranges;
	std::vector<size_t> destinations;
	size_t length = 0;
	for (std::string const& line : lines) {
		if (line.starts_with("seeds: ")) {
			for (std::string const& e : splitString(line.substr(6), ' ')) {
				seeds.push_back(stoul(e));
			}
		}
		else if (line.length() > 1) {
			if (isNumber(line.at(0))) {
				const std::vector<std::string> numbers = splitString(line, ' ');
				destinations.push_back(stoul(numbers.at(0)));
				sources.push_back(stoul(numbers.at(1)));
				ranges.push_back(stoul(numbers.at(2)));
				length++;
			}
		}
		else {
			lengths.push_back(length);
		}
	}
	lengths.push_back(length);
	size_t* deviceSeeds;
	hipMalloc(&deviceSeeds, sizeof(size_t) * seeds.size());
	hipMemcpy(deviceSeeds, seeds.data(), sizeof(size_t) * seeds.size(), hipMemcpyHostToDevice);
	size_t* deviceLengths;
	hipMalloc(&deviceLengths, sizeof(size_t) * lengths.size());
	hipMemcpy(deviceLengths, lengths.data(), sizeof(size_t) * lengths.size(), hipMemcpyHostToDevice);
	size_t* deviceSources;
	hipMalloc(&deviceSources, sizeof(size_t) * sources.size());
	hipMemcpy(deviceSources, sources.data(), sizeof(size_t) * sources.size(), hipMemcpyHostToDevice);
	size_t* deviceRanges;
	hipMalloc(&deviceRanges, sizeof(size_t) * ranges.size());
	hipMemcpy(deviceRanges, ranges.data(), sizeof(size_t) * ranges.size(), hipMemcpyHostToDevice);
	size_t* deviceDestinations;
	hipMalloc(&deviceDestinations, sizeof(size_t) * destinations.size());
	hipMemcpy(deviceDestinations, destinations.data(), sizeof(size_t) * destinations.size(), hipMemcpyHostToDevice);
	size_t* deviceLocations;
	hipMalloc(&deviceLocations, sizeof(size_t) * seeds.size());
	const dim3 grid(1, 1, 1);
	const dim3 block(seeds.size(), 1, 1);
	const size_t shared = 0;
	const hipStream_t stream = 0;
	if (part == 1) {
		findLocations << <grid, block, shared, stream >> > (
			deviceSeeds,
			lengths.size(),
			deviceLengths,
			deviceSources,
			deviceDestinations,
			deviceRanges,
			deviceLocations
			);
		std::cout << "part 1" << std::endl;
	}
	else {
		findLocations << <grid, block, shared, stream >> > (
			deviceSeeds,
			lengths.size(),
			deviceLengths,
			deviceSources,
			deviceDestinations,
			deviceRanges,
			deviceLocations
			);
		std::cout << "part 2" << std::endl;
	}
	size_t* locations = (size_t*)malloc(sizeof(size_t) * seeds.size());
	hipMemcpy(locations, deviceLocations, sizeof(size_t) * seeds.size(), hipMemcpyDeviceToHost);
	size_t result = locations[0];
	for (size_t i = 0; seeds.size() > i; i++) {
		if (result > locations[i]) result = locations[i];
		std::cout << locations[i] << std::endl;
	}
	std::cout << "Result: " << result << std::endl;
	free(locations);
	hipFree(deviceSeeds);
	hipFree(deviceLengths);
	hipFree(deviceSources);
	hipFree(deviceDestinations);
	hipFree(deviceRanges);
	hipFree(deviceLocations);
}
