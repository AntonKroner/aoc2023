#include "hip/hip_runtime.h"
#include "days.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>

static void printDeviceInformation(const int device) {
	struct { int version; const char* name; } architectures[] = {
			{0x30, "Kepler"},
			{0x32, "Kepler"},
			{0x35, "Kepler"},
			{0x37, "Kepler"},
			{0x50, "Maxwell"},
			{0x52, "Maxwell"},
			{0x53, "Maxwell"},
			{0x60, "Pascal"},
			{0x61, "Pascal"},
			{0x62, "Pascal"},
			{0x70, "Volta"},
			{0x72, "Xavier"},
			{0x75, "Turing"},
			{0x80, "Ampere"},
			{0x86, "Ampere"},
			{0x87, "Ampere"},
			{0x89, "Ada"},
			{0x90, "Hopper"},
			{-1, "Graphics Device"} };
	int major = 0;
	int minor = 0;
	hipDeviceGetAttribute(&major, hipDeviceAttributeComputeCapabilityMajor, device);
	hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, device);
	int index = 0;
	const char* result = 0;
	while (architectures[index].version != -1) {
		if (architectures[index].version == ((major << 4) + minor)) {
			result = architectures[index].name;
		}
		index++;
	}
	if (result) {
		printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n",
			device, result, major, minor);
	}
	else {
		printf(
			"Architecture for version %d.%d is undefined."
			"  Default to use %s\n",
			major, minor, architectures[index - 1].name);
	}
}
__global__ static void kernel(char* input) {
	char element = input[threadIdx.x];
	input[threadIdx.x] =
		((((element << 0) >> 24) - 10) << 24) | ((((element << 8) >> 24) - 10) << 16) |
		((((element << 16) >> 24) - 10) << 8) | ((((element << 24) >> 24) - 10) << 0);
}
extern "C" void test(size_t length, char* input) {
	printDeviceInformation(0);
	const size_t size = sizeof(char) * length;
	char* deviceData;
	hipMalloc((void**)&deviceData, size);
	hipMemcpy(deviceData, input, size, hipMemcpyHostToDevice);
	const size_t threads = length / 4;
	dim3 grid(1, 1, 1);
	dim3 block(threads, 1, 1);
	size_t shared = 0;
	hipStream_t stream = 0;
	kernel << <grid, block, shared, stream >> > (deviceData);
	hipMemcpy(input, deviceData, size, hipMemcpyDeviceToHost);
	hipFree(deviceData);
}
