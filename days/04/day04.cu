#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <string>
#include <stdlib.h>

__device__ static int parseInt(const char* string) {
	int result = 0;
	int index = 0;
	while (string[index] >= '0' && string[index] <= '9') {
		result = result * 10 + string[index] - '0';
		index++;
	}
	return result;
}
__device__ static inline bool isNumber(const char character) {
	return character >= '0' && character <= '9';
}
__device__ static int findRowMatches(const char* row) {
	size_t i = 0;
	while (row[i] && row[i] != ':') {
		i++;
	}
	const size_t winnersLength = 10;
	size_t winners[winnersLength];
	size_t w = 0;
	while (row[i] && row[i] != '|') {
		if (isNumber(row[i])) {
			winners[w++] = parseInt(&row[i]);
			while (row[i] && isNumber(row[i])) {
				i++;
			}
		}
		i++;
	}
	const size_t numbersLength = 25;
	size_t numbers[numbersLength];
	size_t n = 0;
	while (row[i] && numbersLength > n) {
		if (isNumber(row[i])) {
			numbers[n++] = parseInt(&row[i]);
			while (row[i] && isNumber(row[i])) {
				i++;
			}
		}
		i++;
	}
	size_t matches = 0;
	for (size_t i = 0; numbersLength > i; i++) {
		for (size_t j = 0; winnersLength > j; j++) {
			if (numbers[i] == winners[j]) {
				matches++;
			}
		}
	}
	return matches ? 1 << (matches - 1) : 0;
}
__global__ static void matchCards(const size_t length, const size_t* lengths, const char* input, size_t* results) {
	//if (threadIdx.x < length) {[]
	const size_t result = findRowMatches(&input[lengths[threadIdx.x]]);
	results[threadIdx.x] = result ? 1 << (result - 1) : 0;
//}
}

//__global__ static void matchCards2(const size_t length, const size_t* lengths, const char* input, size_t* results) {
//	//if (threadIdx.x < length) {[]
//	size_t additional = results[threadIdx.x] = findRowMatches(&input[lengths[threadIdx.x]]);
//	__syncthreads();
//	for (size_t i = threadIdx.x; (threadIdx.x + results[threadIdx.x]) > i && length > i; i++) {
//		size_t matches = results[i];
//		while (matches) {
//			additional += matches;
//			matches--;
//		}
//	}
////}
//}
// This is a very stupid kernel that only computes the correct result *sometimes*. Pls help it get better!!
__global__ static void reduce(const size_t length, const size_t* input, size_t* results) {
	size_t previous = length;
	results[threadIdx.x] = input[threadIdx.x * 2] + ((threadIdx.x * 2 + 1 < previous) ? input[threadIdx.x * 2 + 1] : 0);
	__syncthreads();
	for (size_t remaining = length; remaining > 1; remaining = 1 + ((remaining - 1) / 2)) {
		if (remaining > threadIdx.x) {
			results[threadIdx.x] = results[threadIdx.x * 2] + ((threadIdx.x * 2 + 1 < previous) ? results[threadIdx.x * 2 + 1] : 0);
		}
		previous = remaining;
		__syncthreads();
	}
	__syncthreads();
	if (threadIdx.x == 0) {
		results[threadIdx.x] = results[threadIdx.x * 2] + ((threadIdx.x * 2 + 1 < previous) ? results[threadIdx.x * 2 + 1] : 0);
	}
	__syncthreads();
}
void day04(const size_t part) {
	std::string line;
	std::ifstream input;
	std::vector<std::string> lines;
	input.open("./days/4/input.txt");
	if (input.is_open()) {
		while (std::getline(input, line)) {
			lines.push_back(line);
		}
		input.close();
	}
	else {
		std::cout << "file did not open" << std::endl;
	}
	const size_t length = lines.size();
	std::string flattened;
	std::vector<size_t> lengths;
	lengths.push_back(0);
	for (std::string const& line : lines) {
		flattened += (line + '\0');
		lengths.push_back(flattened.size());
	}
	char* deviceFlattened = 0;
	size_t* deviceLengths = 0;
	hipMalloc(&deviceFlattened, sizeof(char) * flattened.length());
	hipMalloc(&deviceLengths, sizeof(size_t) * lengths.size());
	hipMemcpy(deviceFlattened, flattened.data(), sizeof(char) * flattened.length(), hipMemcpyHostToDevice);
	hipMemcpy(deviceLengths, lengths.data(), sizeof(size_t) * lengths.size(), hipMemcpyHostToDevice);
	size_t* matches;
	hipMalloc(&matches, sizeof(size_t) * length);
	const dim3 grid(1, 1, 1);
	const dim3 block(length, 1, 1);
	const size_t shared = 0;
	const hipStream_t stream = 0;
	if (part == 1) {
		matchCards << <grid, block, shared, stream >> > (length, deviceLengths, deviceFlattened, matches);
		std::cout << "part 1" << std::endl;
	}
	else {
		matchCards << <grid, block, shared, stream >> > (length, deviceLengths, deviceFlattened, matches);
		std::cout << "part 2" << std::endl;
	}
	size_t* results;
	hipMalloc(&results, sizeof(size_t) * 1 + ((length - 1) / 2));
	const dim3 block2(1 + ((length - 1) / 2), 1, 1);
	size_t* hostMatches = (size_t*)malloc(sizeof(size_t) * length);
	hipMemcpy(hostMatches, matches, sizeof(size_t) * length, hipMemcpyDeviceToHost);
	for (size_t i = 0; i < length; i++) {
		std::cout << hostMatches[i] << std::endl;
	}
	free(hostMatches);
	reduce << <grid, block2, shared, stream >> > (length, matches, results);
	hipFree(matches);
	size_t result;
	hipMemcpy(&result, &results[0], sizeof(int), hipMemcpyDeviceToHost);
	std::cout << "result: " << result << std::endl;
	hipFree(results);
	hipFree(deviceFlattened);
	hipFree(deviceLengths);
}
