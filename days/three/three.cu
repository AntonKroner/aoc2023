#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <string>
#include <stdlib.h>

__device__ static int parseInt(const char* string) {
	int result = 0;
	int index = 0;
	while (string[index] >= '0' && string[index] <= '9') {
		result = result * 10 + string[index] - '0';
		index++;
	}
	return result;
}
__device__ static inline bool isSymbol(const char character) {
	return character < '.' || character > '9' || character == '/';
}
__device__ static int sumRow(const size_t length, const char* rows) {
	int result = 0;
	int value = 0;
	for (size_t index = 1; (length - 1) > index; index++) {
		if (!value) {
			value = parseInt(&rows[length + index]);
		}
		if (value) {
			if (rows[length + index] >= '0' && rows[length + index] <= '9') {
				if (
					isSymbol(rows[index - 1]) ||
					isSymbol(rows[index]) ||
					isSymbol(rows[index + 1]) ||
					isSymbol(rows[length + index - 1]) ||
					isSymbol(rows[length + index + 1]) ||
					isSymbol(rows[2 * length + index - 1]) ||
					isSymbol(rows[2 * length + index]) ||
					isSymbol(rows[2 * length + index + 1])
					) {
					result += value;
					value = 0;
					while (rows[length + index] >= '0' && rows[length + index] <= '9') {
						index++;
					}
				}
			}
			else if (rows[length + index] == '.') {
				value = 0;
			}
		}
	}
	return result;
}
__global__ static void sumRows(const ulong2 dimensions, const char* rows, int* results) {
	//if (threadIdx.x < length) {[]
	results[threadIdx.x] = sumRow(dimensions.x, &rows[dimensions.x * threadIdx.x]);
//}
}
__device__ static inline bool isNumber(const char character) {
	return character >= '0' && character <= '9';
}
__device__ static int sumRowRatio(const size_t length, const char* rows) {
	int result = 0;
	for (size_t index = 1; (length - 1) > index; index++) {
		if (rows[length + index] == '*') {
			int ratio = 1;
			size_t found = 0;
			if (isNumber(rows[length + index + 1])) {
				found++;
				ratio = ratio * parseInt(&rows[length + index + 1]);
			}
			if (isNumber(rows[length + index - 1])) {
				found++;
				int numberStartIndex = length + index - 1;
				while (isNumber(rows[numberStartIndex - 1])) {
					numberStartIndex--;
				}
				ratio = ratio * parseInt(&rows[numberStartIndex]);
			}
			if (isNumber(rows[index - 1])) {
				found++;
				int numberStartIndex = index - 1;
				while (isNumber(rows[numberStartIndex - 1])) {
					numberStartIndex--;
				}
				ratio = ratio * parseInt(&rows[numberStartIndex]);
			}
			if (isNumber(rows[index]) && !isNumber(rows[index - 1])) {
				found++;
				ratio = ratio * parseInt(&rows[index]);
			}
			if (isNumber(rows[index + 1]) && !isNumber(rows[index])) {
				found++;
				ratio = ratio * parseInt(&rows[index + 1]);
			}
			if (isNumber(rows[2 * length + index - 1])) {
				found++;
				int numberStartIndex = 2 * length + index - 1;
				while (isNumber(rows[numberStartIndex - 1])) {
					numberStartIndex--;
				}
				ratio = ratio * parseInt(&rows[numberStartIndex]);
			}
			if (isNumber(rows[2 * length + index]) && !isNumber(rows[2 * length + index - 1])) {
				found++;
				ratio = ratio * parseInt(&rows[2 * length + index]);
			}
			if (isNumber(rows[2 * length + index + 1]) && !isNumber(rows[2 * length + index])) {
				found++;
				ratio = ratio * parseInt(&rows[2 * length + index + 1]);
			}
			result = found == 2 ? result + ratio : result;
		}
	}
	return result;
}
__global__ static void findGears(const ulong2 dimensions, const char* rows, int* results) {
	//if (threadIdx.x < length) {[]
	results[threadIdx.x] = sumRowRatio(dimensions.x, &rows[dimensions.x * threadIdx.x]);
//}
}
// This is a very stupid kernel that only computes the correct result *sometimes*. Pls help it get better!!
__global__ static void reduce(const size_t length, const int* games, int* results) {
	results[threadIdx.x] = games[threadIdx.x * 2] + games[threadIdx.x * 2 + 1];
	__syncthreads();
	size_t previous = length;
	for (size_t remaining = length; remaining > 1; remaining = 1 + ((remaining - 1) / 2)) {
		if (remaining > threadIdx.x) {
			results[threadIdx.x] = results[threadIdx.x * 2] + ((threadIdx.x * 2 + 1 < previous) ? results[threadIdx.x * 2 + 1] : 0);
		}
		previous = remaining;
		__syncthreads();
	}
	__syncthreads();
	if (threadIdx.x == 0) {
		results[threadIdx.x] = results[threadIdx.x * 2] + ((threadIdx.x * 2 + 1 < previous) ? results[threadIdx.x * 2 + 1] : 0);
	}
	__syncthreads();
}
void three(const size_t part) {
	std::string line;
	std::ifstream input;
	std::vector<std::string> lines;
	input.open("./days/three/input.txt");
	if (input.is_open()) {
		while (std::getline(input, line)) {
			lines.push_back(line);
		}
		input.close();
	}
	else {
		std::cout << "file did not open" << std::endl;
	}
	const ulong2 dimensions = ulong2(lines.at(0).size() + 2, lines.size() + 2);
	std::string schematic;
	schematic.append(dimensions.x, '.');
	for (std::string const& line : lines) {
		schematic += ('.' + line + '.');
	}
	schematic.append(dimensions.x, '.');
	char* deviceSchematic = 0;
	hipMalloc(&deviceSchematic, sizeof(char) * (dimensions.x) * (dimensions.y));
	hipMemcpy(deviceSchematic, schematic.data(), sizeof(char) * (dimensions.x) * (dimensions.y), hipMemcpyHostToDevice);
	int* rowTotals;
	hipMalloc(&rowTotals, sizeof(int) * dimensions.y - 2);
	const dim3 grid(1, 1, 1);
	const dim3 block(dimensions.y - 2, 1, 1);
	const size_t shared = 0;
	const hipStream_t stream = 0;
	if (part == 1) {
		sumRows << <grid, block, shared, stream >> > (dimensions, deviceSchematic, rowTotals);
		std::cout << "part 1" << std::endl;
	}
	else {
		findGears << <grid, block, shared, stream >> > (dimensions, deviceSchematic, rowTotals);
		std::cout << "part 2" << std::endl;
	}
	int* results;
	hipMalloc(&results, sizeof(int) * dimensions.y / 2);
	const dim3 block2(dimensions.y - 2 / 2, 1, 1);
	int* hostRowTotals = (int*)malloc(sizeof(int) * dimensions.y - 2);
	hipMemcpy(hostRowTotals, rowTotals, sizeof(int) * dimensions.y - 2, hipMemcpyDeviceToHost);
	for (size_t i = 0; i < dimensions.y - 2; i++) {
		std::cout << hostRowTotals[i] << std::endl;
	}
	free(hostRowTotals);
	reduce << <grid, block2, shared, stream >> > (dimensions.y - 2, rowTotals, results);
	//int* hostReduceVector = (int*)malloc(sizeof(int) * dimensions.y - 2);
	//hipMemcpy(hostReduceVector, results, sizeof(int) * dimensions.y - 2, hipMemcpyDeviceToHost);
	//for (size_t i = 0; i < dimensions.y - 2; i++) {
	//	std::cout << hostReduceVector[i] << std::endl;
	//}
	//free(hostReduceVector);
	hipFree(rowTotals);
	size_t result;
	hipMemcpy(&result, &results[0], sizeof(int), hipMemcpyDeviceToHost);
	std::cout << "result: " << result << std::endl;
	hipFree(results);
	hipFree(deviceSchematic);
}
