#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <string>
#include <stdlib.h>

__device__ static int parseInt(const char* string) {
	int result = 0;
	int index = 0;
	while (string[index] >= '0' && string[index] <= '9') {
		result = result * 10 + string[index] - '0';
		index++;
	}
	return result;
}
#define REDMAX 12
#define GREEMMAX 13
#define BLUEMAX 14
__device__ static int game(const char* input) {
	bool success = true;
	int index = 0;
	int amount = 0;
	const int game = parseInt(&input[5]);
	while (input[index] && !(input[index] == ':')) {
		index++;
	}
	index++;
	while (input[index]) {
		if (amount) {
			if (input[index] == 'r') {
				if (amount > REDMAX) {
					success = false;
				}
				amount = 0;
				index += 2;
			}
			if (input[index] == 'g') {
				if (amount > GREEMMAX) {
					success = false;
				}
				amount = 0;
				index += 4;
			}
			if (input[index] == 'b') {
				if (amount > BLUEMAX) {
					success = false;
				}
				amount = 0;
				index += 3;
			}
		}
		else {
			amount = parseInt(&input[index]);
		}
		index++;
	}
	return success ? game : 0;
}
__global__ static void playGames(const size_t length, const size_t* lengths, const char* input, int* results) {
	//if (threadIdx.x < length) {[]
	results[threadIdx.x] = game(&input[lengths[threadIdx.x]]);
//}
}
// This is a very stupid kernel that only computes the correct result *sometimes*. Pls help it get better!!
__global__ static void reduce(const size_t length, const int* games, int* results) {
	results[threadIdx.x] = games[threadIdx.x * 2] + games[threadIdx.x * 2 + 1];
	__syncthreads();
	size_t previous = length;
	for (size_t remaining = length; remaining > 1; remaining = 1 + ((remaining - 1) / 2)) {
		if (remaining > threadIdx.x) {
			results[threadIdx.x] = results[threadIdx.x * 2] + ((threadIdx.x * 2 + 1 < previous) ? results[threadIdx.x * 2 + 1] : 0);
		}
		__syncthreads();
		previous = remaining;
	}
	__syncthreads();
}
void two(const size_t part) {
	std::string line;
	std::ifstream input;
	std::vector<std::string> data;
	input.open("./days/two/input.txt");
	if (input.is_open()) {
		while (std::getline(input, line)) {
			data.push_back(line);
		}
		input.close();
	}
	else {
		std::cout << "file did not open" << std::endl;
	}
	const size_t length = data.size();
	std::string flattened;
	std::vector<size_t> lengths;
	lengths.push_back(0);
	for (std::string const& line : data) {
		flattened += (line + '\0');
		lengths.push_back(flattened.size());
	}
	char* deviceFlattened = 0;
	size_t* deviceLengths = 0;
	hipMalloc(&deviceFlattened, sizeof(char) * flattened.length());
	hipMalloc(&deviceLengths, sizeof(size_t) * lengths.size());
	hipMemcpy(deviceFlattened, flattened.data(), sizeof(char) * flattened.length(), hipMemcpyHostToDevice);
	hipMemcpy(deviceLengths, lengths.data(), sizeof(size_t) * lengths.size(), hipMemcpyHostToDevice);
	int* games;
	hipMalloc(&games, sizeof(int) * length);
	const dim3 grid(1, 1, 1);
	const dim3 block(length, 1, 1);
	const size_t shared = 0;
	const hipStream_t stream = 0;
	if (part == 1) {
		playGames << <grid, block, shared, stream >> > (length, deviceLengths, deviceFlattened, games);
		std::cout << "part 1" << std::endl;
	}
	else {
		playGames << <grid, block, shared, stream >> > (length, deviceLengths, deviceFlattened, games);
		std::cout << "part 2" << std::endl;
	}
	int* results;
	hipMalloc(&results, sizeof(int) * length / 2);
	const dim3 block2(length / 2, 1, 1);
	int* hostCallibrations = (int*)malloc(sizeof(int) * length);
	hipMemcpy(hostCallibrations, games, sizeof(int) * length, hipMemcpyDeviceToHost);
	for (size_t i = 0; i < length; i++) {
		std::cout << hostCallibrations[i] << std::endl;
	}
	free(hostCallibrations);
	reduce << <grid, block2, shared, stream >> > (length, games, results);
	hipFree(games);
	size_t result;
	hipMemcpy(&result, &results[0], sizeof(int), hipMemcpyDeviceToHost);
	std::cout << "result: " << result << std::endl;
	hipFree(results);
	hipFree(deviceFlattened);
	hipFree(deviceLengths);
}
