#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <string>
#include <stdlib.h>
//__device__ static int parseInteger(const char* input) {
//	int result = 0;
//	int index = 0;
//	while (input[index]) {
//		if (input[index] >= '0' || input[index] <= '9') {
//			result = result * 10 + input[index];
//		}
//		index++;
//	}
//	return result;
//}

__device__ int2 findDigits(const char* input) {
	int2 result = make_int2(0, 0);
	int digit = 0;
	int index = 0;
	while (input[index]) {
		if (input[index] >= '0' && input[index] <= '9') {
			digit = input[index] - 48;
			if (result.x == 0) { result.x = digit; }
		}
		index++;
	}
	result.y = digit;
	return result;
}
__global__ void calibrate(const size_t length, const size_t* lengths, const char* input, int* results) {
	//if (threadIdx.x < length) {[]
	const int2 values = findDigits(&input[lengths[threadIdx.x]]);
	results[threadIdx.x] = values.x * 10 + values.y;
//}
}
__global__ void reduce(const size_t length, const int* calibrations, int* results) {
	results[threadIdx.x] = calibrations[threadIdx.x * 2] + calibrations[threadIdx.x * 2 + 1];
	__syncthreads();
	size_t previous = length;
	for (size_t remaining = length; remaining > 1; remaining = 1 + ((remaining - 1) / 2)) {
		if (remaining > threadIdx.x) {
			results[threadIdx.x] = results[threadIdx.x * 2] + ((threadIdx.x * 2 + 1 < previous) ? results[threadIdx.x * 2 + 1] : 0);
		}
		__syncthreads();
		previous = remaining;
	}
	__syncthreads();
}
void one(const size_t part) {
	std::string line;
	std::ifstream input;
	std::vector<std::string> data;
	input.open("./days/one/input.txt");
	if (input.is_open()) {
		while (std::getline(input, line)) {
			data.push_back(line);
		}
		input.close();
	}
	else {
		std::cout << "file did not open" << std::endl;
	}
	const size_t length = data.size();
	std::string flattened;
	std::vector<size_t> lengths;
	lengths.push_back(0);
	for (std::string const& line : data) {
		flattened += (line + '\0');
		lengths.push_back(flattened.size());
	}
	char* deviceFlattened = 0;
	size_t* deviceLengths = 0;
	hipMalloc(&deviceFlattened, sizeof(char) * flattened.length());
	hipMalloc(&deviceLengths, sizeof(size_t) * lengths.size());
	hipMemcpy(deviceFlattened, flattened.data(), sizeof(char) * flattened.length(), hipMemcpyHostToDevice);
	hipMemcpy(deviceLengths, lengths.data(), sizeof(size_t) * lengths.size(), hipMemcpyHostToDevice);
	int* calibrations;
	hipMalloc(&calibrations, sizeof(int) * length);
	const dim3 grid(1, 1, 1);
	const dim3 block(length, 1, 1);
	const size_t shared = 0;
	const hipStream_t stream = 0;
	calibrate << <grid, block, shared, stream >> > (length, deviceLengths, deviceFlattened, calibrations);
	int* results;
	hipMalloc(&results, sizeof(int) * length / 2);
	const dim3 block2(length / 2, 1, 1);
	reduce << <grid, block2, shared, stream >> > (length, calibrations, results);
	hipFree(calibrations);
	size_t result;
	hipMemcpy(&result, &results[0], sizeof(int), hipMemcpyDeviceToHost);
	std::cout << "result: " << result << std::endl;
	hipFree(results);
	hipFree(deviceFlattened);
	hipFree(deviceLengths);
}
