#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <string>

__device__ static int parseInteger(const char* input) {
	int result = 0;
	int index = 0;
	while (input[index]) {
		if (
			input[index] == '0' ||
			input[index] == '1' ||
			input[index] == '2' ||
			input[index] == '3' ||
			input[index] == '4' ||
			input[index] == '5' ||
			input[index] == '6' ||
			input[index] == '7' ||
			input[index] == '8' ||
			input[index] == '9'
			) {
			result = result * 10 + input[index];
		}
		index++;
	}

// return result.
	return result;

}

__device__ static int2 findNumbers(const char* string) {
	int2 result;
	result.x = 0;
	result.y = 0;
	return result;
}
__global__ static void calibrate(const size_t length, const char* input[], int* result) {
	if (threadIdx.x < length) {
		const int2 numbers = findNumbers(input[threadIdx.x]);
		result[threadIdx.x] = numbers.x + numbers.y;
	}
}
void one(const size_t part) {
	std::string line;
	std::ifstream input;
	std::vector<std::string> data;
	input.open("input.txt", std::ios::out);
	if (input.is_open()) {
		while (std::getline(input, line)) {
			data.push_back(line);
		}
		input.close();
	}
	const size_t length = data.size();
	const char** deviceData = 0;
	hipMalloc(&deviceData, sizeof(char*) * data.size());
	hipMemcpy(deviceData, data.data(), sizeof(char*) * data.size(), hipMemcpyHostToDevice);
	int* calibrations = 0;
	hipMalloc(&calibrations, sizeof(int) * data.size());
	const size_t threads = data.size();;
	dim3 grid(1, 1, 1);
	dim3 block(threads, 1, 1);
	size_t shared = 0;
	hipStream_t stream = 0;
	calibrate << <grid, block, shared, stream >> > (length, deviceData, calibrations);
	hipFree(deviceData);
}
