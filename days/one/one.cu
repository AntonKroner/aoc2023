#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_vector_types.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <string>
#include <stdlib.h>
//__device__ static int parseInteger(const char* input) {
//	int result = 0;
//	int index = 0;
//	while (input[index]) {
//		if (input[index] >= '0' || input[index] <= '9') {
//			result = result * 10 + input[index];
//		}
//		index++;
//	}
//	return result;
//}

__device__ static int2 findDigits(const char* input) {
	int2 result = { 0, 0 };
	int digit = 0;
	int index = 0;
	while (input[index]) {
		if (input[index] >= '0' || input[index] <= '9') {
			digit = input[index] - 48;
			if (result.x == 0) { result.x = digit; }
		}
		index++;
	}
	result.y = digit;
	return result;
}
__global__ static void calibrate(const size_t length, const char* input[], int* result) {
	if (threadIdx.x < length) {
		const int2 numbers = findDigits(input[threadIdx.x]);
		result[threadIdx.x] = numbers.x * 10 + numbers.y;
	}
}
void one(const size_t part) {
	std::string line;
	std::ifstream input;
	std::vector<std::string> data;
	std::cout << "aaaa" << std::endl;

	input.open("input.txt");
	if (input.is_open()) {
		std::cout << "file is open" << std::endl;
		while (std::getline(input, line)) {
			data.push_back(line);
			std::cout << line << std::endl;
		}
		input.close();
	}
	else {
		std::cout << "file did not open" << std::endl;

	}
	const size_t length = data.size();
	const char** deviceData = 0;
	hipMalloc(&deviceData, sizeof(char*) * data.size());
	hipMemcpy(deviceData, data.data(), sizeof(char*) * data.size(), hipMemcpyHostToDevice);
	int* calibrations = 0;
	hipMalloc(&calibrations, sizeof(int) * data.size());
	const size_t threads = data.size();;
	dim3 grid(1, 1, 1);
	dim3 block(threads, 1, 1);
	size_t shared = 0;
	hipStream_t stream = 0;
	calibrate << <grid, block, shared, stream >> > (length, deviceData, calibrations);
	int* results = (int*)malloc(sizeof(int) * data.size());
	hipMemcpy(results, calibrations, sizeof(int) * data.size(), hipMemcpyDeviceToHost);
	for (size_t index = 0; data.size() > index; index++) {
		std::cout << results[index] << std::endl;
	}
	free(results);
	hipFree(deviceData);
}
