#include <hip/hip_runtime.h>

#include <hip/hip_vector_types.h>
#include <vector>
#include <iostream>
#include <fstream>
#include <string>
#include <stdlib.h>

__device__ static int2 findDigits(const char* input) {
  int2 result = make_int2(0, 0);
  int digit = 0;
  int index = 0;
  while (input[index]) {
    if (input[index] >= '0' && input[index] <= '9') {
      digit = input[index] - 48;
      if (result.x == 0) {
        result.x = digit;
      }
    }
    index++;
  }
  result.y = digit;
  return result;
}
__global__ static void calibrate1(
        const size_t length,
        const size_t* lengths,
        const char* input,
        int* results
) {
  //if (threadIdx.x < length) {[]
  const int2 values = findDigits(&input[lengths[threadIdx.x]]);
  results[threadIdx.x] = values.x * 10 + values.y;
  //}
}
__device__ int2 static findDigits2(const char* input) {
  int2 result = make_int2(0, 0);
  int digit = 0;
  int index = 0;
  const size_t length = 5;
  char buffer[length] = { '0', '0', '0', '0', '0' };
  while (input[index]) {
    if (input[index] >= '0' && input[index] <= '9') {
      digit = input[index] - 48;
      if (result.x == 0) {
        result.x = digit;
      }
    }
    for (size_t i = 0; i < (length - 1); i++) {
      buffer[i] = buffer[i + 1];
    }
    buffer[4] = input[index];
    if (buffer[2] == 'o' && buffer[3] == 'n' && buffer[4] == 'e') {
      digit = 1;
      if (result.x == 0) {
        result.x = digit;
      }
    }
    else if (buffer[2] == 't' && buffer[3] == 'w' && buffer[4] == 'o') {
      digit = 2;
      if (result.x == 0) {
        result.x = digit;
      }
    }
    else if (buffer[0] == 't' && buffer[1] == 'h' && buffer[2] == 'r' && buffer[3] == 'e' && buffer[4] == 'e') {
      digit = 3;
      if (result.x == 0) {
        result.x = digit;
      }
    }
    else if (buffer[1] == 'f' && buffer[2] == 'o' && buffer[3] == 'u' && buffer[4] == 'r') {
      digit = 4;
      if (result.x == 0) {
        result.x = digit;
      }
    }
    else if (buffer[1] == 'f' && buffer[2] == 'i' && buffer[3] == 'v' && buffer[4] == 'e') {
      digit = 5;
      if (result.x == 0) {
        result.x = digit;
      }
    }
    else if (buffer[2] == 's' && buffer[3] == 'i' && buffer[4] == 'x') {
      digit = 6;
      if (result.x == 0) {
        result.x = digit;
      }
    }
    else if (buffer[0] == 's' && buffer[1] == 'e' && buffer[2] == 'v' && buffer[3] == 'e' && buffer[4] == 'n') {
      digit = 7;
      if (result.x == 0) {
        result.x = digit;
      }
    }
    else if (buffer[0] == 'e' && buffer[1] == 'i' && buffer[2] == 'g' && buffer[3] == 'h' && buffer[4] == 't') {
      digit = 8;
      if (result.x == 0) {
        result.x = digit;
      }
    }
    else if (buffer[1] == 'n' && buffer[2] == 'i' && buffer[3] == 'n' && buffer[4] == 'e') {
      digit = 9;
      if (result.x == 0) {
        result.x = digit;
      }
    }
    index++;
  }
  result.y = digit;
  return result;
}
__global__ static void calibrate2(
        const size_t length,
        const size_t* lengths,
        const char* input,
        int* results
) {
  //if (threadIdx.x < length) {[]
  const int2 values = findDigits2(&input[lengths[threadIdx.x]]);
  results[threadIdx.x] = values.x * 10 + values.y;
  //}
}
// This is a very stupid kernel that only computes the correct result *sometimes*. Pls help it get better!!
__global__ static void reduce(const size_t length, const int* calibrations, int* results) {
  results[threadIdx.x] =
          calibrations[threadIdx.x * 2] + calibrations[threadIdx.x * 2 + 1];
  __syncthreads();
  size_t previous = length;
  for (size_t remaining = length; remaining > 1; remaining = 1 + ((remaining - 1) / 2)) {
    if (remaining > threadIdx.x) {
      results[threadIdx.x] =
              results[threadIdx.x * 2]
              + ((threadIdx.x * 2 + 1 < previous) ? results[threadIdx.x * 2 + 1] : 0);
    }
    __syncthreads();
    previous = remaining;
  }
  __syncthreads();
}
void one(const size_t part) {
  std::string line;
  std::ifstream input;
  std::vector<std::string> data;
  input.open("./days/one/input.txt");
  if (input.is_open()) {
    while (std::getline(input, line)) {
      data.push_back(line);
    }
    input.close();
  }
  else {
    std::cout << "file did not open" << std::endl;
  }
  const size_t length = data.size();
  std::string flattened;
  std::vector<size_t> lengths;
  lengths.push_back(0);
  for (const std::string& line: data) {
    flattened += (line + '\0');
    lengths.push_back(flattened.size());
  }
  char* deviceFlattened = 0;
  size_t* deviceLengths = 0;
  hipMalloc(&deviceFlattened, sizeof(char) * flattened.length());
  hipMalloc(&deviceLengths, sizeof(size_t) * lengths.size());
  hipMemcpy(
          deviceFlattened,
          flattened.data(),
          sizeof(char) * flattened.length(),
          hipMemcpyHostToDevice
  );
  hipMemcpy(
          deviceLengths,
          lengths.data(),
          sizeof(size_t) * lengths.size(),
          hipMemcpyHostToDevice
  );
  int* calibrations;
  hipMalloc(&calibrations, sizeof(int) * length);
  const dim3 grid(1, 1, 1);
  const dim3 block(length, 1, 1);
  const size_t shared = 0;
  const hipStream_t stream = 0;
  if (part == 1) {
    calibrate1<<<grid, block, shared, stream>>>(
            length,
            deviceLengths,
            deviceFlattened,
            calibrations
    );
    std::cout << "part 1" << std::endl;
  }
  else {
    calibrate2<<<grid, block, shared, stream>>>(
            length,
            deviceLengths,
            deviceFlattened,
            calibrations
    );
    std::cout << "part 2" << std::endl;
  }
  int* results;
  hipMalloc(&results, sizeof(int) * length / 2);
  const dim3 block2(length / 2, 1, 1);
  int* hostCallibrations = (int*)malloc(sizeof(int) * length);
  hipMemcpy(hostCallibrations, calibrations, sizeof(int) * length, hipMemcpyDeviceToHost);
  for (size_t i = 0; i < length; i++) {
    std::cout << hostCallibrations[i] << std::endl;
  }
  free(hostCallibrations);
  reduce<<<grid, block2, shared, stream>>>(length, calibrations, results);
  hipFree(calibrations);
  size_t result;
  hipMemcpy(&result, &results[0], sizeof(int), hipMemcpyDeviceToHost);
  std::cout << "result: " << result << std::endl;
  hipFree(results);
  hipFree(deviceFlattened);
  hipFree(deviceLengths);
}
